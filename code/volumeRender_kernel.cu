#include "hip/hip_runtime.h"

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
//#include "cuPrintf.cu"

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d; //to disply inner organ use 1.00
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;

    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}
//unsigned long long int
__global__ void
join(VolumeType *b, VolumeType **p) {

	unsigned long long int width = 220;
	size_t ww = 220;
	unsigned long long int heigth = 128;
	unsigned long long int depth = 234; 

	int bid = blockIdx.x;
	int tid = threadIdx.x;

	int check1 = (bid % 4) / 2;
	int check2 = bid % 2;
	int check3 = bid / 4;
	int check4 = tid / 128;
	int check5 = tid % 128;

	int one = width * heigth;
	int two = width * depth;
	int three = one * depth;

	int i = 0;
	unsigned long long int u;
	unsigned long long int v;

	for (i = 0; i < 8; i++) {
		u = (check3 * 4 * three) + (check1 * 2 * one) + (check2 * width) + (i * 4 * one) + (check5 * 2 * width) + (check4 * 2 * three);
		v = (tid * width) + (i * one) + (check4 * (three / 2));
		memcpy(b + u, p[bid] + v, ww);
	 }
	//cuPrintf("hii");
	# if __CUDA_ARCH__>= 200
		printf("heeeeeeehiii");
	#endif  
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
	printf("plz plz");
    // create 3D array
	hipFree(d_volumeArray);
	hipFree(d_transferFuncArray);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 1.0, 1.0, 0.0, },
        {  0.0, 0.0, 0.0, 0.5, },
        {  1.0, 0.0, 0.0, 0.5, },
        {  1.0, 1.0, 0.0, 0.5, },
        {  1.0, 0.5, 1.0, 0.5, },
        {  1.0, 1.0, 0.5, 0.5, },
        {  1.0, 0.0, 1.0, 0.5, },
        {  0.0, 0.0, 0.0, 0.5, },
        {  0.0, 1.0, 1.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void initCuda1(VolumeType *h_volume, hipExtent volumeSize)
{
    // create 3D array
	
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    
	checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 1.0, 1.0, 0.0, },
        {  0.0, 0.0, 0.0, 0.5, },
        {  1.0, 0.0, 0.0, 0.5, },
        {  1.0, 1.0, 0.0, 0.5, },
        {  1.0, 0.5, 1.0, 0.5, },
        {  1.0, 1.0, 0.5, 0.5, },
        {  1.0, 0.0, 1.0, 0.5, },
        {  0.0, 0.0, 0.0, 0.5, },
        {  0.0, 1.0, 1.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void freeCudaBuffers()
{
    //checkCudaErrors(hipFreeArray(d_volumeArray));
    checkCudaErrors(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                      brightness, transferOffset, transferScale);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}

extern "C"
void call(VolumeType* b, VolumeType ** p) {
	//std::cout << "hii" << std::endl;
	join<<<1, 1>>>(b, p);
	//std::cout << "i m done" << std::endl;
}

__global__
void join1(char* source, char * p, int s) {
	//memcpy(p, source, s);
	//char *h = source;
	//char tem = source;
	//source;
	memcpy((p), (source),2);
}

extern "C"
void call1(char* source, char *p, int s) {
	//std::cout << "hii" << std::endl;
	join1<<<1, 1>>>(source, p, s);
	//std::cout << "i m done" << std::endl;
}

__global__ 
void kernel1(char* BiExponent){
  //for(int i=0; i<500; i++)
     //printf("%c",BiExponent[(1*XDIM)+i]); // I want print line 1
}

extern "C"
void call2(char* BiExponent) {
	//std::cout << "hii" << std::endl;
	kernel1<<< 1, 2 >>> (BiExponent);
	//std::cout << "i m done" << std::endl;
}









#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
